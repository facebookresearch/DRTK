#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <c10/cuda/CUDAGuard.h>
#include <cuda_math_helper.h>
#include <torch/types.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <kernel_utils.h>
#include "edge_grad_kernel.h"

using namespace math;

using at::native::fastAtomicAdd;

template <typename scalar_t>
struct TriInfo {
  typedef typename math::TVec2<scalar_t> scalar2_t;

  const scalar2_t p_0;
  const scalar2_t p_1;
  const scalar2_t v_01;
  const scalar2_t v_02;
  const scalar2_t v_12;
  const scalar_t denominator;
};

template <typename scalar_t>
__device__ bool pix_in_tri(const TriInfo<scalar_t>& tri, const int x, const int y) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;

  if (tri.denominator != 0.f) {
    const scalar2_t p = {(scalar_t)x, (scalar_t)y};

    const scalar2_t vp0p = p - tri.p_0;
    const scalar2_t vp1p = p - tri.p_1;

    scalar3_t bary = scalar3_t({
        vp1p.y * tri.v_12.x - vp1p.x * tri.v_12.y,
        vp0p.x * tri.v_02.y - vp0p.y * tri.v_02.x,
        vp0p.y * tri.v_01.x - vp0p.x * tri.v_01.y,
    });
    bary *= sign(tri.denominator);

    const bool on_edge_or_inside = (bary.x >= 0.f) && (bary.y >= 0.f) && (bary.z >= 0.f);

    bool on_edge_0 = bary.x == 0.f;
    bool on_edge_1 = bary.y == 0.f;
    bool on_edge_2 = bary.z == 0.f;

    const bool is_top_left_0 = (tri.denominator > 0)
        ? (tri.v_12.y < 0.f || tri.v_12.y == 0.0f && tri.v_12.x > 0.f)
        : (tri.v_12.y > 0.f || tri.v_12.y == 0.0f && tri.v_12.x < 0.f);
    const bool is_top_left_1 = (tri.denominator > 0)
        ? (tri.v_02.y > 0.f || tri.v_02.y == 0.0f && tri.v_02.x < 0.f)
        : (tri.v_02.y < 0.f || tri.v_02.y == 0.0f && tri.v_02.x > 0.f);
    const bool is_top_left_2 = (tri.denominator > 0)
        ? (tri.v_01.y < 0.f || tri.v_01.y == 0.0f && tri.v_01.x > 0.f)
        : (tri.v_01.y > 0.f || tri.v_01.y == 0.0f && tri.v_01.x < 0.f);

    const bool is_top_left_or_inside = on_edge_or_inside &&
        !(on_edge_0 && !is_top_left_0 || on_edge_1 && !is_top_left_1 ||
          on_edge_2 && !is_top_left_2);
    return is_top_left_or_inside;
  }
  return false;
}

template <typename scalar_t, typename index_t>
__device__ TriInfo<scalar_t>
get_tri_info(const scalar_t* v_ptr, index_t v_sV, index_t v_sC, int3 vi) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  const scalar2_t p_0 = {v_ptr[v_sV * vi.x + v_sC * 0], v_ptr[v_sV * vi.x + v_sC * 1]};
  const scalar2_t p_1 = {v_ptr[v_sV * vi.y + v_sC * 0], v_ptr[v_sV * vi.y + v_sC * 1]};
  const scalar2_t p_2 = {v_ptr[v_sV * vi.z + v_sC * 0], v_ptr[v_sV * vi.z + v_sC * 1]};

  const scalar2_t v_01 = p_1 - p_0;
  const scalar2_t v_02 = p_2 - p_0;
  const scalar2_t v_12 = p_2 - p_1;

  const scalar_t denominator = v_01.x * v_02.y - v_01.y * v_02.x;

  return {p_0, p_1, v_01, v_02, v_12, denominator};
}

template <typename scalar_t, typename index_t>
__device__ math::TVec3<scalar_t>
get_tri_normal(const scalar_t* v_ptr, index_t v_sV, index_t v_sC, int3 vi) {
  typedef typename math::TVec3<scalar_t> scalar3_t;
  const scalar3_t p_0 = {
      v_ptr[v_sV * vi.x + v_sC * 0], v_ptr[v_sV * vi.x + v_sC * 1], v_ptr[v_sV * vi.x + v_sC * 2]};
  const scalar3_t p_1 = {
      v_ptr[v_sV * vi.y + v_sC * 0], v_ptr[v_sV * vi.y + v_sC * 1], v_ptr[v_sV * vi.y + v_sC * 2]};
  const scalar3_t p_2 = {
      v_ptr[v_sV * vi.z + v_sC * 0], v_ptr[v_sV * vi.z + v_sC * 1], v_ptr[v_sV * vi.z + v_sC * 2]};
  return normalize(cross(p_0 - p_2, p_1 - p_0));
}

template <typename scalar_t>
__device__ math::TVec2<scalar_t> get_db_dp(
    const math::TVec2<scalar_t>& n_varying_,
    const math::TVec2<scalar_t>& n_fixed_) {
  /*
      Computes derivative of the point position with respect to edge displacement
      Args:
        - n_varying_: Projection of the normal of the movable triangle onto the plane of
     consideration (XZ or YZ) N x 3 x H x W.
        - n_fixed_:  Projection of the normal of the fixed triangle onto the plane of consideration
     (XZ or YZ) N x 3 x H x W.
     Please refer to the paper "Rasterized Edge Gradients: Handling Discontinuities Differentiably"
     for details.
  */
  typedef typename math::TVec2<scalar_t> scalar2_t;

  const auto n_varying = normalize(n_varying_);
  const auto n_fixed = normalize(n_fixed_);
  const scalar2_t b = {-n_fixed.y, n_fixed.x};
  const auto b_dot_varyingg = dot(b, n_varying);
  return b.x / epsclamp(b_dot_varyingg) * n_varying;
}

template <typename scalar_t, typename index_t>
__device__ math::TVec3<scalar_t> load_vec3_if_valid(
    const scalar_t* __restrict ptr,
    index_t stride,
    bool valid,
    const math::TVec3<scalar_t>& def) {
  if (valid) {
    return {ptr[0 * stride], ptr[1 * stride], ptr[2 * stride]};
  }
  return def;
}

template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(256)
__global__ void edge_grad_backward_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> v_pix,
    TensorInfo<scalar_t, index_t> img,
    TensorInfo<int32_t, index_t> index_img,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<scalar_t, index_t> grad_output,
    TensorInfo<scalar_t, index_t> grad_v_pix_img,
    const index_t memory_span) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;

  const index_t v_pix_sN = v_pix.strides[0];
  const index_t v_pix_sV = v_pix.strides[1];
  const index_t v_pix_sC = v_pix.strides[2];

  const index_t C = img.sizes[1];
  const index_t H = img.sizes[2];
  const index_t W = img.sizes[3];
  const index_t V = v_pix.sizes[1];

  const index_t index_img_sN = index_img.strides[0];
  const index_t index_img_sH = index_img.strides[1];
  const index_t index_img_sW = index_img.strides[2];

  const index_t img_sN = img.strides[0];
  const index_t img_sC = img.strides[1];
  const index_t img_sH = img.strides[2];
  const index_t img_sW = img.strides[3];

  const index_t grad_output_sN = grad_output.strides[0];
  const index_t grad_output_sC = grad_output.strides[1];
  const index_t grad_output_sH = grad_output.strides[2];
  const index_t grad_output_sW = grad_output.strides[3];

  const index_t grad_v_pix_img_sN = grad_v_pix_img.strides[0];
  const index_t grad_v_pix_img_sC = grad_v_pix_img.strides[1];
  const index_t grad_v_pix_img_sH = grad_v_pix_img.strides[2];
  const index_t grad_v_pix_img_sW = grad_v_pix_img.strides[3];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sV = vi.strides[1];
  const index_t vi_sF = vi.strides[2];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t x = index % W;
    const index_t y = (index / W) % H;
    const index_t n = index / (H * W);

    if (x < (W - 1) && y < (H - 1)) {
      //   center-right-down (CRD)
      //
      //   *--------*--------*
      //   | center |  right |
      //   | (0, 0) | (1, 0) |
      //   *--------*--------*
      //   | down   |
      //   | (0, 1) |
      //   *--------*

      // Computing indicator variables
      // BEGIN
      // triangle indices of CRD pixels
      const int32_t* __restrict index_img_ptr = index_img.data + n * index_img_sN;
      const int32_t center_index = index_img_ptr[(y + 0) * index_img_sH + (x + 0) * index_img_sW];
      const int32_t right_index = index_img_ptr[(y + 0) * index_img_sH + (x + 1) * index_img_sW];
      const int32_t down_index = index_img_ptr[(y + 1) * index_img_sH + (x + 0) * index_img_sW];

      // valid mask
      const bool c_valid = (center_index >= 0);
      const bool r_valid = (right_index >= 0);
      const bool d_valid = (down_index >= 0);

      // vertex indices of triangles of CRD pixels
      // 0,0,0 - if not valid
      const int3 vi_pt_center = load_vec3_if_valid<int32_t, index_t>(
          vi.data + n * vi_sN + center_index * vi_sV, vi_sF, c_valid, {0, 0, 0});
      const int3 vi_pt_right = load_vec3_if_valid<int32_t, index_t>(
          vi.data + n * vi_sN + right_index * vi_sV, vi_sF, r_valid, {0, 0, 0});
      const int3 vi_pt_down = load_vec3_if_valid<int32_t, index_t>(
          vi.data + n * vi_sN + down_index * vi_sV, vi_sF, d_valid, {0, 0, 0});

      // center <-> right differ
      const bool lr_diff = (center_index != right_index);
      // center <-> down differ
      const bool ud_diff = (center_index != down_index);

      // if horizontal pair (vertical edge) composed of two triangles
      const bool x_both_valid = c_valid && r_valid;
      // if vertical pair (horizontal edge) composed of two triangles
      const bool y_both_valid = c_valid && d_valid;

      // Get CRD triangle info
      const scalar_t* __restrict v_pix_ptr = v_pix.data + n * v_pix_sN;
      const auto tri_center = get_tri_info(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_center);
      const auto tri_right = get_tri_info(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_right);
      const auto tri_down = get_tri_info(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_down);

      // Compute indicators of edge type
      const bool center_pix_in_right_tri = lr_diff && x_both_valid && pix_in_tri(tri_right, x, y);
      const bool right_pix_in_center_tri =
          lr_diff && x_both_valid && pix_in_tri(tri_center, x + 1, y);
      const bool center_pix_in_down_tri = ud_diff && y_both_valid && pix_in_tri(tri_down, x, y);
      const bool down_pix_in_center_tri =
          ud_diff && y_both_valid && pix_in_tri(tri_center, x, y + 1);

      // Overlap flags
      const bool l_over_r = center_pix_in_right_tri && (!right_pix_in_center_tri);
      const bool r_over_l = right_pix_in_center_tri && (!center_pix_in_right_tri);
      const bool u_over_d = center_pix_in_down_tri && (!down_pix_in_center_tri);
      const bool d_over_u = down_pix_in_center_tri && (!center_pix_in_down_tri);

      // Intersection flags
      const bool horiz_int = center_pix_in_right_tri && right_pix_in_center_tri;
      const bool vert_int = center_pix_in_down_tri && down_pix_in_center_tri;

      // Intersection flags
      const bool horiz_adjacent =
          lr_diff && x_both_valid && (!center_pix_in_right_tri && !right_pix_in_center_tri);
      const bool vert_adjacent =
          ud_diff && y_both_valid && (!center_pix_in_down_tri && !down_pix_in_center_tri);

      // END

      // Compute image gradient dot output gradient from backward
      // This is computed regardless of the edge type as long as there is an edge (lr_diff or
      // ud_diff) BEGIN
      const scalar_t* __restrict img_ptr = img.data + img_sN * n;
      const scalar_t* __restrict grad_output_ptr = grad_output.data + grad_output_sN * n;

      scalar_t grad_dot_x = 0.f;
      scalar_t grad_dot_y = 0.f;
      if (lr_diff) {
        const scalar_t* __restrict img_ptr_right = img_ptr + y * img_sH + (x + 1) * img_sW;
        const scalar_t* __restrict img_ptr_center = img_ptr + y * img_sH + (x + 0) * img_sW;
        const scalar_t* __restrict grad_output_ptr_right =
            grad_output_ptr + y * grad_output_sH + (x + 1) * grad_output_sW;
        const scalar_t* __restrict grad_output_ptr_center =
            grad_output_ptr + y * grad_output_sH + (x + 0) * grad_output_sW;
        for (size_t c = 0; c < C; ++c) {
          grad_dot_x += (img_ptr_right[c * img_sC] - img_ptr_center[c * img_sC]) *
              (0.5f *
               (grad_output_ptr_right[c * grad_output_sC] +
                grad_output_ptr_center[c * grad_output_sC]));
        }
      }
      if (ud_diff) {
        const scalar_t* __restrict img_ptr_down = img_ptr + (y + 1) * img_sH + x * img_sW;
        const scalar_t* __restrict img_ptr_center = img_ptr + (y + 0) * img_sH + x * img_sW;
        const scalar_t* __restrict grad_output_ptr_down =
            grad_output_ptr + (y + 1) * grad_output_sH + x * grad_output_sW;
        const scalar_t* __restrict grad_output_ptr_center =
            grad_output_ptr + (y + 0) * grad_output_sH + x * grad_output_sW;
        for (size_t c = 0; c < C; ++c) {
          grad_dot_y += (img_ptr_down[c * img_sC] - img_ptr_center[c * img_sC]) *
              (0.5f *
               (grad_output_ptr_down[c * grad_output_sC] +
                grad_output_ptr_center[c * grad_output_sC]));
        }
      }
      // END

      scalar3_t grad_v_pix_center = {0.f, 0.f, 0.f};
      scalar3_t grad_v_pix_right = {0.f, 0.f, 0.f};
      scalar3_t grad_v_pix_down = {0.f, 0.f, 0.f};

      const scalar3_t center_normal = get_tri_normal(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_center);
      const scalar3_t right_normal = get_tri_normal(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_right);
      const scalar3_t down_normal = get_tri_normal(v_pix_ptr, v_pix_sV, v_pix_sC, vi_pt_down);

      if (!horiz_int) {
        grad_v_pix_center.x += (!c_valid || r_over_l || horiz_adjacent) ? 0.f : grad_dot_x;
        grad_v_pix_right.x += (!r_valid || l_over_r || horiz_adjacent) ? 0.f : grad_dot_x;
      } else {
        // Center triangle moves, right fixed.
        scalar2_t dbx_dp = get_db_dp<scalar_t>(
            {center_normal.x, center_normal.z}, {right_normal.x, right_normal.z});
        grad_v_pix_center.x += grad_dot_x * dbx_dp.x;
        grad_v_pix_center.z += grad_dot_x * dbx_dp.y;

        // Center triangle fixed, right moves.
        dbx_dp = get_db_dp<scalar_t>(
            {right_normal.x, right_normal.z}, {center_normal.x, center_normal.z});
        grad_v_pix_right.x += grad_dot_x * dbx_dp.x;
        grad_v_pix_right.z += grad_dot_x * dbx_dp.y;
      }

      if (!vert_int) {
        grad_v_pix_center.y += (!c_valid || d_over_u || vert_adjacent) ? 0.f : grad_dot_y;
        grad_v_pix_down.y += (!d_valid || u_over_d || vert_adjacent) ? 0.f : grad_dot_y;
      } else {
        // Center triangle moves, lower fixed.
        scalar2_t dby_dp =
            get_db_dp<scalar_t>({center_normal.y, center_normal.z}, {down_normal.y, down_normal.z});
        grad_v_pix_center.y += grad_dot_y * dby_dp.x;
        grad_v_pix_center.z += grad_dot_y * dby_dp.x;

        // Center triangle fixed, lower moves.
        dby_dp =
            get_db_dp<scalar_t>({down_normal.y, down_normal.z}, {center_normal.y, center_normal.z});
        grad_v_pix_down.y += grad_dot_y * dby_dp.x;
        grad_v_pix_down.z += grad_dot_y * dby_dp.x;
      }

      // Writing grads out
      // BEGIN
      scalar_t* __restrict grad_v_pix_img_ptr = grad_v_pix_img.data + grad_v_pix_img_sN * n;

      // center
      auto* ptr_c = grad_v_pix_img_ptr + (y + 0) * grad_v_pix_img_sH + (x + 0) * grad_v_pix_img_sW;
      atomicAdd(ptr_c + 0 * grad_v_pix_img_sC, -grad_v_pix_center.x);
      atomicAdd(ptr_c + 1 * grad_v_pix_img_sC, -grad_v_pix_center.y);
      atomicAdd(ptr_c + 2 * grad_v_pix_img_sC, -grad_v_pix_center.z);

      // right
      auto* ptr_r = grad_v_pix_img_ptr + (y + 0) * grad_v_pix_img_sH + (x + 1) * grad_v_pix_img_sW;
      atomicAdd(ptr_r + 0 * grad_v_pix_img_sC, -grad_v_pix_right.x);
      atomicAdd(ptr_r + 1 * grad_v_pix_img_sC, -grad_v_pix_right.y);
      atomicAdd(ptr_r + 2 * grad_v_pix_img_sC, -grad_v_pix_right.z);

      // down
      auto* ptr_d = grad_v_pix_img_ptr + (y + 1) * grad_v_pix_img_sH + (x + 0) * grad_v_pix_img_sW;
      atomicAdd(ptr_d + 0 * grad_v_pix_img_sC, -grad_v_pix_down.x);
      atomicAdd(ptr_d + 1 * grad_v_pix_img_sC, -grad_v_pix_down.y);
      atomicAdd(ptr_d + 2 * grad_v_pix_img_sC, -grad_v_pix_down.z);
      // END
    }
  }
}

template <typename scalar_t, typename index_type>
void edge_grad_estimator_cuda_backward_(
    const int64_t count,
    const torch::Tensor& v_pix,
    const torch::Tensor& img,
    const torch::Tensor& index_img,
    const torch::Tensor& vi,
    const torch::Tensor& grad_outputs,
    const torch::Tensor& grad_v_pix_img) {
  edge_grad_backward_kernel<scalar_t, index_type>
      <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
          static_cast<index_type>(count),
          getTensorInfo<scalar_t, index_type>(v_pix),
          getTensorInfo<scalar_t, index_type>(img),
          getTensorInfo<int32_t, index_type>(index_img),
          getTensorInfo<int32_t, index_type>(vi),
          getTensorInfo<scalar_t, index_type>(grad_outputs),
          getTensorInfo<scalar_t, index_type>(grad_v_pix_img),
          grad_v_pix_img.numel());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

torch::Tensor edge_grad_estimator_cuda_backward(
    const torch::Tensor& v_pix,
    const torch::Tensor& img,
    const torch::Tensor& index_img,
    const torch::Tensor& vi,
    const torch::Tensor& grad_outputs) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(img));

  const auto N = img.sizes()[0];
  const auto C = img.sizes()[1];
  const auto H = img.sizes()[2];
  const auto W = img.sizes()[3];
  const auto V = v_pix.sizes()[1];
  const auto count = N * H * W;

  auto grad_v_pix_img = torch::zeros({N, 3, H, W}, v_pix.options());

  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES(v_pix.scalar_type(), "edge_grad_estimator_kernel", [&] {
      if (at::native::canUse32BitIndexMath(v_pix) && at::native::canUse32BitIndexMath(img) &&
          at::native::canUse32BitIndexMath(index_img) && at::native::canUse32BitIndexMath(vi) &&
          at::native::canUse32BitIndexMath(grad_outputs) &&
          at::native::canUse32BitIndexMath(grad_v_pix_img)) {
        edge_grad_estimator_cuda_backward_<scalar_t, int>(
            count, v_pix, img, index_img, vi, grad_outputs, grad_v_pix_img);
      } else {
        edge_grad_estimator_cuda_backward_<scalar_t, int64_t>(
            count, v_pix, img, index_img, vi, grad_outputs, grad_v_pix_img);
      }
    });
  }
  return grad_v_pix_img;
}
