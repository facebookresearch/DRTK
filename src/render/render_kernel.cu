#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <c10/cuda/CUDAGuard.h>
#include <cuda_math_helper.h>
#include <torch/types.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include "render_kernel.h"

#include <kernel_utils.h>

using namespace math;

using at::native::fastAtomicAdd;

template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(256)
__global__ void render_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> v,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int32_t, index_t> index_img,
    TensorInfo<scalar_t, index_t> depth_img,
    TensorInfo<scalar_t, index_t> bary_img) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;

  const index_t H = bary_img.sizes[2];
  const index_t W = bary_img.sizes[3];
  const index_t V = v.sizes[1];

  const index_t v_sN = v.strides[0];
  const index_t v_sV = v.strides[1];
  const index_t v_sC = v.strides[2];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sV = vi.strides[1];
  const index_t vi_sF = vi.strides[2];

  const index_t index_img_sN = index_img.strides[0];
  const index_t index_img_sH = index_img.strides[1];
  const index_t index_img_sW = index_img.strides[2];

  const index_t depth_img_sN = depth_img.strides[0];
  const index_t depth_img_sH = depth_img.strides[1];
  const index_t depth_img_sW = depth_img.strides[2];

  const index_t bary_img_sN = bary_img.strides[0];
  const index_t bary_img_sB = bary_img.strides[1];
  const index_t bary_img_sH = bary_img.strides[2];
  const index_t bary_img_sW = bary_img.strides[3];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t w = index % W;
    const index_t h = (index / W) % H;
    const index_t n = index / (H * W);

    const int32_t tr_index = index_img.data[n * index_img_sN + h * index_img_sH + w * index_img_sW];
    scalar_t* __restrict bary_img_ptr =
        bary_img.data + bary_img_sN * n + bary_img_sH * h + bary_img_sW * w;
    scalar_t* __restrict depth_img_ptr =
        depth_img.data + depth_img_sN * n + depth_img_sH * h + depth_img_sW * w;

    if (tr_index != -1) {
      const int32_t* __restrict vi_ptr = vi.data + n * vi_sN + tr_index * vi_sV;
      const int32_t vi_0 = vi_ptr[0 * vi_sF];
      const int32_t vi_1 = vi_ptr[1 * vi_sF];
      const int32_t vi_2 = vi_ptr[2 * vi_sF];

      assert(vi_0 < V && vi_1 < V && vi_2 < V);

      const scalar_t* __restrict v_ptr = v.data + n * v_sN;
      const scalar2_t p_0 = {v_ptr[v_sV * vi_0 + v_sC * 0], v_ptr[v_sV * vi_0 + v_sC * 1]};
      const scalar2_t p_1 = {v_ptr[v_sV * vi_1 + v_sC * 0], v_ptr[v_sV * vi_1 + v_sC * 1]};
      const scalar2_t p_2 = {v_ptr[v_sV * vi_2 + v_sC * 0], v_ptr[v_sV * vi_2 + v_sC * 1]};

      const scalar3_t p_012_z = {
          v_ptr[v_sV * vi_0 + v_sC * 2],
          v_ptr[v_sV * vi_1 + v_sC * 2],
          v_ptr[v_sV * vi_2 + v_sC * 2]};

      const scalar2_t v_01 = p_1 - p_0;
      const scalar2_t v_02 = p_2 - p_0;
      const scalar_t denominator = epsclamp((v_01.x * v_02.y - v_01.y * v_02.x));

      const scalar2_t vp0p = {w - p_0.x, h - p_0.y};

      const scalar2_t bary_12_pre = scalar2_t{
          (vp0p.x * v_02.y - vp0p.y * v_02.x),
          (vp0p.y * v_01.x - vp0p.x * v_01.y),
      };
      const scalar2_t bary_12 = bary_12_pre / denominator;
      scalar3_t bary = {scalar_t(1.0) - bary_12.x - bary_12.y, bary_12.x, bary_12.y};

      const scalar3_t p_012_z_eps = epsclamp(p_012_z);
      const scalar3_t d_inv = 1.0 / p_012_z_eps;

      const scalar_t depth_inverse = dot(d_inv, bary);
      const scalar_t depth = 1.0f / epsclamp(depth_inverse);

      const scalar3_t bary_3D = d_inv * bary * depth;
      bary_img_ptr[bary_img_sB * 0] = bary_3D.x;
      bary_img_ptr[bary_img_sB * 1] = bary_3D.y;
      bary_img_ptr[bary_img_sB * 2] = bary_3D.z;
      *depth_img_ptr = depth;
    } else {
      bary_img_ptr[bary_img_sB * 0] = scalar_t(0);
      bary_img_ptr[bary_img_sB * 1] = scalar_t(0);
      bary_img_ptr[bary_img_sB * 2] = scalar_t(0);
      *depth_img_ptr = scalar_t(0);
    }
  }
}

template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(256)
__global__ void render_backward_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> v,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int32_t, index_t> index_img,
    TensorInfo<scalar_t, index_t> grad_depth_img,
    TensorInfo<scalar_t, index_t> grad_bary_img,
    TensorInfo<scalar_t, index_t> grad_v,
    const index_t memory_span) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;

  const index_t H = grad_bary_img.sizes[2];
  const index_t W = grad_bary_img.sizes[3];
  const index_t V = v.sizes[1];

  const index_t v_sN = v.strides[0];
  const index_t v_sV = v.strides[1];
  const index_t v_sC = v.strides[2];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sV = vi.strides[1];
  const index_t vi_sF = vi.strides[2];

  const index_t index_img_sN = index_img.strides[0];
  const index_t index_img_sH = index_img.strides[1];
  const index_t index_img_sW = index_img.strides[2];

  const index_t grad_depth_img_sN = grad_depth_img.strides[0];
  const index_t grad_depth_img_sH = grad_depth_img.strides[1];
  const index_t grad_depth_img_sW = grad_depth_img.strides[2];

  const index_t grad_bary_img_sN = grad_bary_img.strides[0];
  const index_t grad_bary_img_sB = grad_bary_img.strides[1];
  const index_t grad_bary_img_sH = grad_bary_img.strides[2];
  const index_t grad_bary_img_sW = grad_bary_img.strides[3];

  const index_t grad_v_sN = grad_v.strides[0];
  const index_t grad_v_sV = grad_v.strides[1];
  const index_t grad_v_sC = grad_v.strides[2];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t w = index % W;
    const index_t h = (index / W) % H;
    const index_t n = index / (H * W);

    const int32_t tr_index = index_img.data[n * index_img_sN + h * index_img_sH + w * index_img_sW];
    const scalar_t* __restrict grad_bary_img_ptr =
        grad_bary_img.data + grad_bary_img_sN * n + grad_bary_img_sH * h + grad_bary_img_sW * w;
    const scalar_t* __restrict grad_depth_img_ptr =
        grad_depth_img.data + grad_depth_img_sN * n + grad_depth_img_sH * h + grad_depth_img_sW * w;

    scalar_t* __restrict grad_v_ptr = grad_v.data + grad_v_sN * n;

    if (tr_index != -1) {
      const int32_t* __restrict vi_ptr = vi.data + n * vi_sN + tr_index * vi_sV;
      const int32_t vi_0 = vi_ptr[0 * vi_sF];
      const int32_t vi_1 = vi_ptr[1 * vi_sF];
      const int32_t vi_2 = vi_ptr[2 * vi_sF];

      assert(vi_0 < V && vi_1 < V && vi_2 < V);

      const scalar_t* __restrict v_ptr = v.data + n * v_sN;
      const scalar2_t p_0 = {v_ptr[v_sV * vi_0 + v_sC * 0], v_ptr[v_sV * vi_0 + v_sC * 1]};
      const scalar2_t p_1 = {v_ptr[v_sV * vi_1 + v_sC * 0], v_ptr[v_sV * vi_1 + v_sC * 1]};
      const scalar2_t p_2 = {v_ptr[v_sV * vi_2 + v_sC * 0], v_ptr[v_sV * vi_2 + v_sC * 1]};

      const scalar3_t p_012_z = {
          v_ptr[v_sV * vi_0 + v_sC * 2],
          v_ptr[v_sV * vi_1 + v_sC * 2],
          v_ptr[v_sV * vi_2 + v_sC * 2]};

      const scalar2_t v_01 = p_1 - p_0;
      const scalar2_t v_02 = p_2 - p_0;
      const scalar_t _denominator = v_01.x * v_02.y - v_01.y * v_02.x;
      const scalar_t denominator = epsclamp(_denominator);
      const bool denominator_clamped = denominator != _denominator;

      const scalar2_t vp0p = {w - p_0.x, h - p_0.y};

      const scalar2_t bary_12_pre = scalar2_t{
          vp0p.x * v_02.y - vp0p.y * v_02.x,
          vp0p.y * v_01.x - vp0p.x * v_01.y,
      };
      const scalar2_t bary_12 = bary_12_pre / denominator;
      scalar3_t bary = {scalar_t(1.0) - bary_12.x - bary_12.y, bary_12.x, bary_12.y};

      const scalar3_t p_012_z_eps = epsclamp(p_012_z);

      const bool z0_clamped = p_012_z_eps.x != p_012_z.x;
      const bool z1_clamped = p_012_z_eps.y != p_012_z.y;
      const bool z2_clamped = p_012_z_eps.z != p_012_z.z;

      const scalar3_t d_inv = 1.0 / p_012_z_eps;

      const scalar_t depth_inverse = dot(d_inv, bary);
      const scalar_t depth_inverse_eps = epsclamp(depth_inverse);
      const bool depth_inverse_clamped = depth_inverse_eps != depth_inverse;
      const scalar_t depth = 1.0f / depth_inverse_eps;

      const scalar3_t dL_bary_3D = {
          grad_bary_img_ptr[grad_bary_img_sB * 0],
          grad_bary_img_ptr[grad_bary_img_sB * 1],
          grad_bary_img_ptr[grad_bary_img_sB * 2]};
      const scalar_t dL_depth = *grad_depth_img_ptr + dot(dL_bary_3D * d_inv, bary);

      const scalar_t dL_depth_inverse =
          depth_inverse_clamped ? 0.f : (-dL_depth / (depth_inverse * depth_inverse));
      const scalar3_t dL_d_inv = dL_bary_3D * bary * depth + dL_depth_inverse * bary;
      const scalar3_t dL_p_012_z = -dL_d_inv / (p_012_z_eps * p_012_z_eps);

      fastAtomicAdd(
          grad_v_ptr,
          grad_v_sV * vi_0 + grad_v_sC * 2,
          memory_span,
          z0_clamped ? 0.f : dL_p_012_z.x,
          true);
      fastAtomicAdd(
          grad_v_ptr,
          grad_v_sV * vi_1 + grad_v_sC * 2,
          memory_span,
          z1_clamped ? 0.f : dL_p_012_z.y,
          true);
      fastAtomicAdd(
          grad_v_ptr,
          grad_v_sV * vi_2 + grad_v_sC * 2,
          memory_span,
          z2_clamped ? 0.f : dL_p_012_z.z,
          true);

      const scalar3_t dL_bary = dL_bary_3D * d_inv * depth + dL_depth_inverse * d_inv;
      const scalar2_t dL_bary_12 = {-dL_bary.x + dL_bary.y, -dL_bary.x + dL_bary.z};
      const scalar2_t dL_bary_pre = dL_bary_12 / denominator;

      const scalar_t dL_denominator = denominator_clamped ? 0.f : -dot(dL_bary_pre, bary_12);

      const scalar2_t dL_vp0p = {
          dL_bary_pre.x * v_02.y - dL_bary_pre.y * v_01.y,
          -dL_bary_pre.x * v_02.x + dL_bary_pre.y * v_01.x};

      const scalar2_t dL_v_02 = {
          -dL_bary_pre.x * vp0p.y - dL_denominator * v_01.y,
          dL_bary_pre.x * vp0p.x + dL_denominator * v_01.x};
      const scalar2_t dL_v_01 = {
          dL_bary_pre.y * vp0p.y + dL_denominator * v_02.y,
          -dL_bary_pre.y * vp0p.x - dL_denominator * v_02.x};

      const scalar2_t dL_p0 = -dL_v_02 - dL_v_01 - dL_vp0p;
      const scalar2_t dL_p1 = dL_v_01;
      const scalar2_t dL_p2 = dL_v_02;

      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_0 + grad_v_sC * 0, memory_span, dL_p0.x, true);
      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_0 + grad_v_sC * 1, memory_span, dL_p0.y, true);
      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_1 + grad_v_sC * 0, memory_span, dL_p1.x, true);
      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_1 + grad_v_sC * 1, memory_span, dL_p1.y, true);
      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_2 + grad_v_sC * 0, memory_span, dL_p2.x, true);
      fastAtomicAdd(grad_v_ptr, grad_v_sV * vi_2 + grad_v_sC * 1, memory_span, dL_p2.y, true);
    }
  }
}

std::vector<torch::Tensor>
render_cuda(const torch::Tensor& v, const torch::Tensor& vi, const torch::Tensor& index_img) {
  TORCH_CHECK(
      v.defined() && vi.defined() && index_img.defined(),
      "render(): expected all inputs to be defined");
  auto v_opt = v.options();
  auto vi_opt = vi.options();
  auto index_img_opt = index_img.options();
  TORCH_CHECK(
      (v.device() == vi.device()) && (v.device() == index_img.device()) && (v.is_cuda()),
      "render(): expected all inputs to be on same cuda device");
  TORCH_CHECK(
      v.is_floating_point(),
      "render(): expected v to have floating point type, but v has ",
      v.dtype());
  TORCH_CHECK(
      vi.dtype() == torch::kInt32,
      "render(): expected vi to have int32 type, but vi has ",
      vi.dtype());
  TORCH_CHECK(
      index_img.dtype() == torch::kInt32,
      "render(): expected index_img to have int32 type, but index_img has ",
      index_img.dtype());
  TORCH_CHECK(
      v.layout() == torch::kStrided && vi.layout() == torch::kStrided &&
          index_img.layout() == torch::kStrided,
      "render(): expected all inputs to have torch.strided layout");
  TORCH_CHECK(
      (v.dim() == 3) && (vi.dim() == 3) && (index_img.dim() == 3),
      "render(): expected v.ndim == 3, vi.ndim == 3, index_img.ndim == 3, "
      "but got v with sizes ",
      v.sizes(),
      " and vi with sizes ",
      vi.sizes(),
      " and index_img with sizes ",
      index_img.sizes());
  TORCH_CHECK(
      v.size(0) == index_img.size(0),
      "render(): expected v and index_img to have same batch size, "
      "but got v with sizes ",
      v.sizes(),
      " and index_img with sizes ",
      index_img.sizes());
  TORCH_CHECK(
      vi.size(0) == v.size(0),
      "rasterize(): expected first dim of vi to match first dim of v but got ",
      v.size(0),
      " in first dim of v, and ",
      vi.size(0),
      " in the first dim of vi");
  TORCH_CHECK(
      v.size(2) == 3 && vi.size(2) == 3,
      "render(): expected third dim of v to be of size 3, and third dim of vi to be of size 3, but got ",
      v.size(2),
      " in the third dim of v, and ",
      vi.size(2),
      " in the third dim of vi");

  const at::cuda::OptionalCUDAGuard device_guard(device_of(v));

  auto N = v.size(0);
  auto H = index_img.size(1);
  auto W = index_img.size(2);
  int64_t count = N * H * W;

  auto depth_img = at::empty({N, H, W}, v.options());
  auto bary_img = at::empty({N, 3, H, W}, v.options());

  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES(v.scalar_type(), "render_kernel", [&] {
      if (at::native::canUse32BitIndexMath(v) && at::native::canUse32BitIndexMath(bary_img) &&
          at::native::canUse32BitIndexMath(depth_img) &&
          at::native::canUse32BitIndexMath(index_img) && at::native::canUse32BitIndexMath(vi)) {
        typedef int index_type;

        render_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(v),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(depth_img),
                getTensorInfo<scalar_t, index_type>(bary_img));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        typedef int64_t index_type;

        render_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(v),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(depth_img),
                getTensorInfo<scalar_t, index_type>(bary_img));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return {depth_img, bary_img};
}

torch::Tensor render_cuda_backward(
    const torch::Tensor& v,
    const torch::Tensor& vi,
    const torch::Tensor& index_img,
    const torch::Tensor& grad_depth_img,
    const torch::Tensor& grad_bary_img) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(v));

  auto N = v.size(0);
  auto V = v.size(1);
  auto C = v.size(2);
  auto H = index_img.size(1);
  auto W = index_img.size(2);
  int64_t count = N * H * W;

  auto grad_v = at::zeros({N, V, C}, v.options());

  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES(v.scalar_type(), "interpolate_kernel", [&] {
      if (at::native::canUse32BitIndexMath(v) && at::native::canUse32BitIndexMath(grad_bary_img) &&
          at::native::canUse32BitIndexMath(grad_v) && at::native::canUse32BitIndexMath(index_img) &&
          at::native::canUse32BitIndexMath(grad_depth_img) &&
          at::native::canUse32BitIndexMath(vi)) {
        typedef int index_type;

        render_backward_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(v),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(grad_depth_img),
                getTensorInfo<scalar_t, index_type>(grad_bary_img),
                getTensorInfo<scalar_t, index_type>(grad_v),
                grad_v.numel());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        typedef int64_t index_type;

        render_backward_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(v),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(grad_depth_img),
                getTensorInfo<scalar_t, index_type>(grad_bary_img),
                getTensorInfo<scalar_t, index_type>(grad_v),
                grad_v.numel());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return grad_v;
}
