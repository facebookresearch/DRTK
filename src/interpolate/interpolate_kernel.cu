#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <c10/cuda/CUDAGuard.h>
#include <cuda_math_helper.h>
#include <torch/types.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <hipcub/hipcub.hpp>

#include <kernel_utils.h>

using at::native::fastAtomicAdd;

template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(256)
__global__ void interpolate_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> vert_attributes,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int32_t, index_t> index_img,
    TensorInfo<scalar_t, index_t> bary_img,
    TensorInfo<scalar_t, index_t> out_img) {
  const index_t C = vert_attributes.sizes[2];
  const index_t H = bary_img.sizes[2];
  const index_t W = bary_img.sizes[3];

  const index_t vert_attributes_sN = vert_attributes.strides[0];
  const index_t vert_attributes_sV = vert_attributes.strides[1];
  const index_t vert_attributes_sC = vert_attributes.strides[2];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sV = vi.strides[1];
  const index_t vi_sF = vi.strides[2];

  const index_t index_img_sN = index_img.strides[0];
  const index_t index_img_sH = index_img.strides[1];
  const index_t index_img_sW = index_img.strides[2];

  const index_t bary_img_sN = bary_img.strides[0];
  const index_t bary_img_sB = bary_img.strides[1];
  const index_t bary_img_sH = bary_img.strides[2];
  const index_t bary_img_sW = bary_img.strides[3];

  const index_t out_img_sN = out_img.strides[0];
  const index_t out_img_sC = out_img.strides[1];
  const index_t out_img_sH = out_img.strides[2];
  const index_t out_img_sW = out_img.strides[3];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t w = index % W;
    const index_t h = (index / W) % H;
    const index_t n = index / (H * W);

    const int32_t tr_index = index_img.data[n * index_img_sN + h * index_img_sH + w * index_img_sW];
    scalar_t* __restrict out_ptr = out_img.data + out_img_sN * n + out_img_sH * h + out_img_sW * w;

    if (tr_index != -1) {
      const int32_t* __restrict vi_ptr = vi.data + n * vi_sN + tr_index * vi_sV;
      const int32_t vi_0 = vi_ptr[0 * vi_sF];
      const int32_t vi_1 = vi_ptr[1 * vi_sF];
      const int32_t vi_2 = vi_ptr[2 * vi_sF];

      const scalar_t* __restrict vert_ptr = vert_attributes.data + vert_attributes_sN * n;
      const scalar_t* vert_0_ptr = vert_ptr + vert_attributes_sV * vi_0;
      const scalar_t* vert_1_ptr = vert_ptr + vert_attributes_sV * vi_1;
      const scalar_t* vert_2_ptr = vert_ptr + vert_attributes_sV * vi_2;

      const scalar_t* __restrict bary_ptr =
          bary_img.data + bary_img_sN * n + bary_img_sH * h + bary_img_sW * w;
      const scalar_t bary_0 = bary_ptr[0 * bary_img_sB];
      const scalar_t bary_1 = bary_ptr[1 * bary_img_sB];
      const scalar_t bary_2 = bary_ptr[2 * bary_img_sB];

      for (int i = 0; i < C; ++i) {
        scalar_t v0 = vert_0_ptr[i * vert_attributes_sC];
        scalar_t v1 = vert_1_ptr[i * vert_attributes_sC];
        scalar_t v2 = vert_2_ptr[i * vert_attributes_sC];
        out_ptr[out_img_sC * i] = v0 * bary_0 + v1 * bary_1 + v2 * bary_2;
      }
    } else {
      for (int i = 0; i < C; ++i) {
        const scalar_t v[2] = {(w * 2.0f + 1.0f) / W - 1.0f, (h * 2.0f + 1.0f) / H - 1.0f};
        out_ptr[out_img_sC * i] = v[i % 2];
      }
    }
  }
}

template <typename scalar_t, typename index_t, bool bary_img_requires_grad, bool vert_requires_grad>
C10_LAUNCH_BOUNDS_1(256)
__global__ void interpolate_backward_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> grad_out,
    TensorInfo<scalar_t, index_t> vert_attributes,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int32_t, index_t> index_img,
    TensorInfo<scalar_t, index_t> bary_img,
    TensorInfo<scalar_t, index_t> vert_attributes_grad,
    TensorInfo<scalar_t, index_t> bary_img_grad,
    const index_t memory_span) {
  index_t C = vert_attributes.sizes[2];
  index_t H = bary_img.sizes[2];
  index_t W = bary_img.sizes[3];

  index_t vert_attributes_sN = vert_attributes.strides[0];
  index_t vert_attributes_sV = vert_attributes.strides[1];
  index_t vert_attributes_sC = vert_attributes.strides[2];

  index_t vert_attributes_grad_sN = vert_attributes_grad.strides[0];
  index_t vert_attributes_grad_sV = vert_attributes_grad.strides[1];
  index_t vert_attributes_grad_sC = vert_attributes_grad.strides[2];

  index_t vi_sN = vi.strides[0];
  index_t vi_sV = vi.strides[1];
  index_t vi_sF = vi.strides[2];

  index_t index_img_sN = index_img.strides[0];
  index_t index_img_sH = index_img.strides[1];
  index_t index_img_sW = index_img.strides[2];

  index_t bary_img_sN = bary_img.strides[0];
  index_t bary_img_sB = bary_img.strides[1];
  index_t bary_img_sH = bary_img.strides[2];
  index_t bary_img_sW = bary_img.strides[3];

  index_t bary_img_grad_sN = bary_img_grad.strides[0];
  index_t bary_img_grad_sB = bary_img_grad.strides[1];
  index_t bary_img_grad_sH = bary_img_grad.strides[2];
  index_t bary_img_grad_sW = bary_img_grad.strides[3];

  index_t grad_out_sN = grad_out.strides[0];
  index_t grad_out_sC = grad_out.strides[1];
  index_t grad_out_sH = grad_out.strides[2];
  index_t grad_out_sW = grad_out.strides[3];

  int64_t index = blockIdx.x * blockDim.x + threadIdx.x;

  constexpr int warp_size = 32;
  int lane = threadIdx.x % warp_size;

  __shared__ typename hipcub::WarpReduce<scalar_t>::TempStorage temp_storage_0;
  __shared__ typename hipcub::WarpReduce<scalar_t>::TempStorage temp_storage_1;
  __shared__ typename hipcub::WarpReduce<scalar_t>::TempStorage temp_storage_2;

  {
    const index_t w = index % W;
    const index_t h = (index / W) % H;
    const index_t n = index / (H * W);

    int32_t tr_index = -1;

    if (index < nthreads)
      tr_index = index_img.data[n * index_img_sN + h * index_img_sH + w * index_img_sW];
    const scalar_t* __restrict grad_out_ptr =
        grad_out.data + grad_out_sN * n + grad_out_sH * h + grad_out_sW * w;
    scalar_t* __restrict bary_grad_ptr =
        bary_img_grad.data + bary_img_grad_sN * n + bary_img_grad_sH * h + bary_img_grad_sW * w;

    bool thread_is_used = tr_index != -1;

    // True if at least one thread in the warp is used.
    bool warp_is_used = __any_sync(0xFFFFFFFFU, thread_is_used);

    if (warp_is_used) {
      int32_t vi_0 = -1, vi_1 = -1, vi_2 = -1;
      if (thread_is_used) {
        const int32_t* __restrict vi_ptr = vi.data + n * vi_sN + tr_index * vi_sV;
        vi_0 = vi_ptr[0 * vi_sF];
        vi_1 = vi_ptr[1 * vi_sF];
        vi_2 = vi_ptr[2 * vi_sF];
      }
      unsigned m = 0xFFFFFFFFU;
      int vi_0_head = (__shfl_up_sync(m, vi_0, 1) != vi_0) || (lane == 0);
      int vi_0_tail = (__shfl_down_sync(m, vi_0, 1) != vi_0) || (lane == (warp_size - 1));
      int vi_1_head = (__shfl_up_sync(m, vi_1, 1) != vi_1) || (lane == 0);
      int vi_1_tail = (__shfl_down_sync(m, vi_1, 1) != vi_1) || (lane == (warp_size - 1));
      int vi_2_head = (__shfl_up_sync(m, vi_2, 1) != vi_2) || (lane == 0);
      int vi_2_tail = (__shfl_down_sync(m, vi_2, 1) != vi_2) || (lane == (warp_size - 1));

      const scalar_t* __restrict vert_ptr = vert_attributes.data + vert_attributes_sN * n;
      const scalar_t* vert_0_ptr = vert_ptr + vert_attributes_sV * vi_0;
      const scalar_t* vert_1_ptr = vert_ptr + vert_attributes_sV * vi_1;
      const scalar_t* vert_2_ptr = vert_ptr + vert_attributes_sV * vi_2;

      scalar_t* __restrict vert_grad_ptr = vert_attributes_grad.data + vert_attributes_grad_sN * n;
      scalar_t* vert_0_grad_ptr = vert_grad_ptr + vert_attributes_grad_sV * vi_0;
      scalar_t* vert_1_grad_ptr = vert_grad_ptr + vert_attributes_grad_sV * vi_1;
      scalar_t* vert_2_grad_ptr = vert_grad_ptr + vert_attributes_grad_sV * vi_2;

      const scalar_t* __restrict bary_ptr =
          bary_img.data + bary_img_sN * n + bary_img_sH * h + bary_img_sW * w;
      scalar_t bary_0, bary_1, bary_2;

      if (thread_is_used && vert_requires_grad) {
        bary_0 = bary_ptr[0 * bary_img_sB];
        bary_1 = bary_ptr[1 * bary_img_sB];
        bary_2 = bary_ptr[2 * bary_img_sB];
      }

      auto bary_0_grad = scalar_t(0.);
      auto bary_1_grad = scalar_t(0.);
      auto bary_2_grad = scalar_t(0.);

      for (int i = 0; i < C; ++i) {
        scalar_t g_out = grad_out_ptr[i * grad_out_sC];
        if (thread_is_used && bary_img_requires_grad) {
          scalar_t v0 = vert_0_ptr[i * vert_attributes_sC];
          scalar_t v1 = vert_1_ptr[i * vert_attributes_sC];
          scalar_t v2 = vert_2_ptr[i * vert_attributes_sC];

          bary_0_grad += g_out * v0;
          bary_1_grad += g_out * v1;
          bary_2_grad += g_out * v2;
        }

        if (vert_requires_grad) {
          scalar_t grad_v_0 =
              hipcub::WarpReduce<scalar_t>(temp_storage_0).TailSegmentedSum(g_out * bary_0, vi_0_tail);
          scalar_t grad_v_1 =
              hipcub::WarpReduce<scalar_t>(temp_storage_1).TailSegmentedSum(g_out * bary_1, vi_1_tail);
          scalar_t grad_v_2 =
              hipcub::WarpReduce<scalar_t>(temp_storage_2).TailSegmentedSum(g_out * bary_2, vi_2_tail);

          __syncthreads();

          if (vi_0_head && thread_is_used)
            fastAtomicAdd(
                vert_0_grad_ptr, i * vert_attributes_grad_sC, memory_span, grad_v_0, true);
          if (vi_1_head && thread_is_used)
            fastAtomicAdd(
                vert_1_grad_ptr, i * vert_attributes_grad_sC, memory_span, grad_v_1, true);
          if (vi_2_head && thread_is_used)
            fastAtomicAdd(
                vert_2_grad_ptr, i * vert_attributes_grad_sC, memory_span, grad_v_2, true);
        }
      }
      if (bary_img_requires_grad) {
        if (thread_is_used) {
          bary_grad_ptr[0 * bary_img_grad_sB] = bary_0_grad;
          bary_grad_ptr[1 * bary_img_grad_sB] = bary_1_grad;
          bary_grad_ptr[2 * bary_img_grad_sB] = bary_2_grad;
        } else {
          bary_grad_ptr[0 * bary_img_grad_sB] = scalar_t(0.);
          bary_grad_ptr[1 * bary_img_grad_sB] = scalar_t(0.);
          bary_grad_ptr[2 * bary_img_grad_sB] = scalar_t(0.);
        }
      }
    } else if ((index < nthreads) && bary_img_requires_grad) {
      bary_grad_ptr[0 * bary_img_grad_sB] = scalar_t(0.);
      bary_grad_ptr[1 * bary_img_grad_sB] = scalar_t(0.);
      bary_grad_ptr[2 * bary_img_grad_sB] = scalar_t(0.);
    }
  }
}

torch::Tensor interpolate_cuda(
    const torch::Tensor& vert_attributes,
    const torch::Tensor& vi,
    const torch::Tensor& index_img,
    const torch::Tensor& bary_img) {
  TORCH_CHECK(
      vert_attributes.defined() && vi.defined() && index_img.defined() && bary_img.defined(),
      "interpolate(): expected all inputs to be defined");
  auto vert_attributes_opt = vert_attributes.options();
  auto vi_opt = vi.options();
  auto index_img_opt = index_img.options();
  auto bary_img_opt = bary_img.options();
  TORCH_CHECK(
      (vert_attributes.device() == vi.device()) &&
          (vert_attributes.device() == index_img.device()) &&
          (vert_attributes.device() == bary_img.device()),
      "interpolate(): expected all inputs to be on same device");
  TORCH_CHECK(
      vert_attributes.dtype() == bary_img.dtype(),
      "interpolate(): expected vert_attributes and bary_img to have same dtype, but vert_attributes has ",
      vert_attributes.dtype(),
      " and bary_img has ",
      bary_img.dtype());
  TORCH_CHECK(
      vert_attributes.is_floating_point(),
      "interpolate(): expected vert_attributes to have floating point type, but v has ",
      vert_attributes.dtype());
  TORCH_CHECK(
      vi.dtype() == torch::kInt32,
      "interpolate(): expected vi to have int32 type, but vi has ",
      vi.dtype());
  TORCH_CHECK(
      index_img.dtype() == torch::kInt32,
      "interpolate(): expected index_img to have int32 type, but index_img has ",
      index_img.dtype());
  TORCH_CHECK(
      vert_attributes.layout() == torch::kStrided && vi.layout() == torch::kStrided &&
          index_img.layout() == torch::kStrided && bary_img.layout() == torch::kStrided,
      "interpolate(): expected all inputs to have torch.strided layout");
  TORCH_CHECK(
      (vert_attributes.dim() == 3) && (vi.dim() == 3) && (index_img.dim() == 3) &&
          (bary_img.dim() == 4),
      "interpolate(): expected vert_attributes.ndim == 3, vi.ndim == 3, index_img.ndim == 3, bary_img.ndim == 4, "
      "but got vert_attributes with sizes ",
      vert_attributes.sizes(),
      " and vi with sizes ",
      vi.sizes(),
      " and index_img with sizes ",
      index_img.sizes(),
      " and bary_img with sizes ",
      bary_img.sizes());
  TORCH_CHECK(
      vert_attributes.size(0) == index_img.size(0) && vert_attributes.size(0) == bary_img.size(0),
      "interpolate(): expected vert_attributes, index_img and bary_img to have same batch size, "
      "but got vert_attributes with sizes ",
      vert_attributes.sizes(),
      " and index_img with sizes ",
      index_img.sizes(),
      " and bary_img with sizes ",
      bary_img.sizes());
  TORCH_CHECK(
      vi.size(2) == 3 && bary_img.size(1) == 3,
      "interpolate(): expected last dim of vi to be of size 3, and second dim of bary_img to be of size 3, but got ",
      vi.size(2),
      " in the last dim of vi, and ",
      bary_img.size(1),
      " in the second dim of bary_img");
  TORCH_CHECK(
      vi.size(0) == vert_attributes.size(0),
      "interpolate(): expected vi to have same first dimension as vert_atrributes, but got ",
      vi.size(0),
      " in the first dim of vi, and ",
      vert_attributes.size(0),
      " in the first dim of vert_attributes");
  TORCH_CHECK(
      index_img.size(1) == bary_img.size(2) && index_img.size(2) == bary_img.size(3),
      "interpolate(): expected H and W dims of index_img and bary_img to match");

  const at::cuda::OptionalCUDAGuard device_guard(device_of(vert_attributes));

  auto N = vert_attributes.size(0);
  auto V = vert_attributes.size(1);
  auto C = vert_attributes.size(2);
  auto H = bary_img.size(2);
  auto W = bary_img.size(3);
  int64_t count = N * H * W;

  auto output = at::empty({N, C, H, W}, vert_attributes.options());

  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES(vert_attributes.scalar_type(), "interpolate_kernel", [&] {
      if (at::native::canUse32BitIndexMath(vert_attributes) &&
          at::native::canUse32BitIndexMath(bary_img) &&
          at::native::canUse32BitIndexMath(index_img) && at::native::canUse32BitIndexMath(vi)) {
        typedef int index_type;

        interpolate_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(vert_attributes),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(bary_img),
                getTensorInfo<scalar_t, index_type>(output));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        typedef int64_t index_type;

        interpolate_kernel<scalar_t, index_type>
            <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                static_cast<index_type>(count),
                getTensorInfo<scalar_t, index_type>(vert_attributes),
                getTensorInfo<int32_t, index_type>(vi),
                getTensorInfo<int32_t, index_type>(index_img),
                getTensorInfo<scalar_t, index_type>(bary_img),
                getTensorInfo<scalar_t, index_type>(output));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return output;
}

template <typename scalar_t, typename index_t, bool bary_img_requires_grad, bool vert_requires_grad>
void _interpolate_cuda_backward(
    int64_t count,
    const torch::Tensor& grad_out,
    const torch::Tensor& vert_attributes,
    const torch::Tensor& vi,
    const torch::Tensor& index_img,
    const torch::Tensor& bary_img,
    const torch::Tensor& vert_attributes_grad,
    const torch::Tensor& bary_img_grad) {
  interpolate_backward_kernel<scalar_t, index_t, bary_img_requires_grad, vert_requires_grad>
      <<<GET_BLOCKS(count, 256), 256, 0, at::cuda::getCurrentCUDAStream()>>>(
          static_cast<index_t>(count),
          getTensorInfo<scalar_t, index_t>(grad_out),
          getTensorInfo<scalar_t, index_t>(vert_attributes),
          getTensorInfo<int32_t, index_t>(vi),
          getTensorInfo<int32_t, index_t>(index_img),
          getTensorInfo<scalar_t, index_t>(bary_img),
          vert_requires_grad ? getTensorInfo<scalar_t, index_t>(vert_attributes_grad)
                             : TensorInfo<scalar_t, index_t>({nullptr, {0}, {0}, 0}),
          bary_img_requires_grad ? getTensorInfo<scalar_t, index_t>(bary_img_grad)
                                 : TensorInfo<scalar_t, index_t>({nullptr, {0}, {0}, 0}),
          vert_attributes_grad.numel());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename scalar_t, typename index_t>
void _interpolate_cuda_backward(
    int64_t count,
    const torch::Tensor& grad_out,
    const torch::Tensor& vert_attributes,
    const torch::Tensor& vi,
    const torch::Tensor& index_img,
    const torch::Tensor& bary_img,
    const torch::Tensor& vert_attributes_grad,
    const torch::Tensor& bary_img_grad,
    bool bary_img_requires_grad,
    bool vert_requires_grad) {
  if (bary_img_requires_grad && vert_requires_grad)
    _interpolate_cuda_backward<scalar_t, index_t, true, true>(
        count,
        grad_out,
        vert_attributes,
        vi,
        index_img,
        bary_img,
        vert_attributes_grad,
        bary_img_grad);
  else if (bary_img_requires_grad)
    _interpolate_cuda_backward<scalar_t, index_t, true, false>(
        count,
        grad_out,
        vert_attributes,
        vi,
        index_img,
        bary_img,
        vert_attributes_grad,
        bary_img_grad);
  else if (vert_requires_grad)
    _interpolate_cuda_backward<scalar_t, index_t, false, true>(
        count,
        grad_out,
        vert_attributes,
        vi,
        index_img,
        bary_img,
        vert_attributes_grad,
        bary_img_grad);
}

std::tuple<torch::Tensor, torch::Tensor> interpolate_cuda_backward(
    const torch::Tensor& grad_out,
    const torch::Tensor& vert_attributes,
    const torch::Tensor& vi,
    const torch::Tensor& index_img,
    const torch::Tensor& bary_img) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(vert_attributes));

  auto N = vert_attributes.size(0);
  auto V = vert_attributes.size(1);
  auto C = vert_attributes.size(2);
  auto H = bary_img.size(2);
  auto W = bary_img.size(3);
  int64_t count = N * H * W;

  bool bary_img_requires_grad = bary_img.requires_grad();
  bool vert_requires_grad = vert_attributes.requires_grad();

  auto vert_attributes_grad =
      vert_requires_grad ? at::zeros({N, V, C}, vert_attributes.options()) : torch::Tensor();
  auto bary_img_grad =
      bary_img_requires_grad ? at::empty({N, 3, H, W}, bary_img.options()) : torch::Tensor();

  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES(vert_attributes.scalar_type(), "interpolate_kernel", [&] {
      if (at::native::canUse32BitIndexMath(vert_attributes) &&
          at::native::canUse32BitIndexMath(bary_img) &&
          at::native::canUse32BitIndexMath(index_img) && at::native::canUse32BitIndexMath(vi)) {
        _interpolate_cuda_backward<scalar_t, int>(
            count,
            grad_out,
            vert_attributes,
            vi,
            index_img,
            bary_img,
            vert_attributes_grad,
            bary_img_grad,
            bary_img_requires_grad,
            vert_requires_grad);
      } else {
        _interpolate_cuda_backward<scalar_t, int64_t>(
            count,
            grad_out,
            vert_attributes,
            vi,
            index_img,
            bary_img,
            vert_attributes_grad,
            bary_img_grad,
            bary_img_requires_grad,
            vert_requires_grad);
      }
    });
  }
  return std::make_tuple(vert_attributes_grad, bary_img_grad);
}
