#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <c10/cuda/CUDAGuard.h>
#include <cuda_math_helper.h>
#include <grid_utils.h>
#include <torch/types.h>

#include <limits>

#include "rasterize_kernel.h"

#include <kernel_utils.h>

using namespace math;

template <typename scalar_t, typename index_t>
__global__ void rasterize_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> v,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int64_t, index_t> packed_index_depth_img) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;
  typedef typename math::TVec4<scalar_t> scalar4_t;

  const index_t H = packed_index_depth_img.sizes[1];
  const index_t W = packed_index_depth_img.sizes[2];
  const index_t V = v.sizes[1];
  const index_t n_prim = vi.sizes[1];

  const index_t index_sN = packed_index_depth_img.strides[0];
  const index_t index_sH = packed_index_depth_img.strides[1];
  const index_t index_sW = packed_index_depth_img.strides[2];

  const index_t v_sN = v.strides[0];
  const index_t v_sV = v.strides[1];
  const index_t v_sC = v.strides[2];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sF = vi.strides[1];
  const index_t vi_sI = vi.strides[2];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t n = index / n_prim;
    const index_t id = index % n_prim;

    const int32_t* __restrict vi_ptr = vi.data + vi_sN * n + vi_sF * id;
    const int32_t vi_0 = (int32_t)(((uint32_t)vi_ptr[vi_sI * 0]) & 0x0FFFFFFFU);
    const int32_t vi_1 = vi_ptr[vi_sI * 1];
    const int32_t vi_2 = vi_ptr[vi_sI * 2];

    assert(vi_0 < V && vi_1 < V && vi_2 < V);

    // Skip degenerate triangles. Useful for padding of vi buffer for batched rasterization
    bool triangle_is_degenerate = (vi_0 == vi_1) && (vi_1 == vi_2);

    const scalar_t* __restrict v_ptr = v.data + n * v_sN;
    const scalar2_t p_0 = {v_ptr[v_sV * vi_0 + v_sC * 0], v_ptr[v_sV * vi_0 + v_sC * 1]};
    const scalar2_t p_1 = {v_ptr[v_sV * vi_1 + v_sC * 0], v_ptr[v_sV * vi_1 + v_sC * 1]};
    const scalar2_t p_2 = {v_ptr[v_sV * vi_2 + v_sC * 0], v_ptr[v_sV * vi_2 + v_sC * 1]};

    const scalar3_t p_012_z = {
        v_ptr[v_sV * vi_0 + v_sC * 2],
        v_ptr[v_sV * vi_1 + v_sC * 2],
        v_ptr[v_sV * vi_2 + v_sC * 2]};

    const scalar2_t min_p = math::min(math::min(p_0, p_1), p_2);
    const scalar2_t max_p = math::max(math::max(p_0, p_1), p_2);

    const bool all_z_greater_0 = math::all_greater(p_012_z, {1e-8f, 1e-8f, 1e-8f});
    const bool in_canvas = math::all_less_or_eq(min_p, {(scalar_t)(W - 1), (scalar_t)(H - 1)}) &&
        math::all_greater(max_p, {0.f, 0.f});

    if (all_z_greater_0 && in_canvas && ~triangle_is_degenerate) {
      const scalar2_t v_01 = p_1 - p_0;
      const scalar2_t v_02 = p_2 - p_0;
      const scalar2_t v_12 = p_2 - p_1;

      const scalar_t denominator = v_01.x * v_02.y - v_01.y * v_02.x;

      if (denominator != 0.f) {
        // Compute triangle bounds with extra border.
        int min_x = max(0, int(min_p.x));
        int min_y = max(0, int(min_p.y));

        int max_x = min((int)W - 1, int(max_p.x) + 1);
        int max_y = min((int)H - 1, int(max_p.y) + 1);

        // Loop over pixels inside triangle bbox.
        for (int y = min_y; y <= max_y; ++y) {
          for (int x = min_x; x <= max_x; ++x) {
            const scalar2_t p = {(scalar_t)x, (scalar_t)y};

            const scalar2_t vp0p = p - p_0;
            const scalar2_t vp1p = p - p_1;

            scalar3_t bary = scalar3_t({
                vp1p.y * v_12.x - vp1p.x * v_12.y,
                vp0p.x * v_02.y - vp0p.y * v_02.x,
                vp0p.y * v_01.x - vp0p.x * v_01.y,
            });
            bary *= sign(denominator);

            const bool on_edge_or_inside = (bary.x >= 0.f) && (bary.y >= 0.f) && (bary.z >= 0.f);

            bool on_edge_0 = bary.x == 0.f;
            bool on_edge_1 = bary.y == 0.f;
            bool on_edge_2 = bary.z == 0.f;

            const bool is_top_left_0 = (denominator > 0)
                ? (v_12.y < 0.f || v_12.y == 0.0f && v_12.x > 0.f)
                : (v_12.y > 0.f || v_12.y == 0.0f && v_12.x < 0.f);
            const bool is_top_left_1 = (denominator > 0)
                ? (v_02.y > 0.f || v_02.y == 0.0f && v_02.x < 0.f)
                : (v_02.y < 0.f || v_02.y == 0.0f && v_02.x > 0.f);
            const bool is_top_left_2 = (denominator > 0)
                ? (v_01.y < 0.f || v_01.y == 0.0f && v_01.x > 0.f)
                : (v_01.y > 0.f || v_01.y == 0.0f && v_01.x < 0.f);

            const bool is_top_left_or_inside = on_edge_or_inside &&
                !(on_edge_0 && !is_top_left_0 || on_edge_1 && !is_top_left_1 ||
                  on_edge_2 && !is_top_left_2);

            if (is_top_left_or_inside) {
              bary /= abs(denominator);

              // interpolate inverse depth linearly
              const scalar3_t d_inv = 1.0 / epsclamp(p_012_z);
              const scalar_t depth_inverse = dot(d_inv, bary);
              const scalar_t depth = 1.0f / epsclamp(depth_inverse);

              const unsigned long long packed_val =
                  (static_cast<unsigned long long>(__float_as_uint(depth)) << 32u) |
                  static_cast<unsigned long long>(id);
              atomicMin(
                  reinterpret_cast<unsigned long long*>(packed_index_depth_img.data) +
                      index_sN * n + index_sH * y + index_sW * x,
                  packed_val);
            }
          }
        }
      }
    }
  }
}

template <typename scalar_t>
__device__ inline void get_line(
    const math::TVec2<scalar_t>& p1,
    const math::TVec2<scalar_t>& p2,
    scalar_t& a,
    scalar_t& b,
    scalar_t& c) {
  a = p1.y - p2.y;
  b = p2.x - p1.x;
  c = p1.x * p2.y - p2.x * p1.y;
}

template <typename scalar_t>
__device__ inline bool is_point_in_segment(
    const math::TVec2<scalar_t>& p1,
    const math::TVec2<scalar_t>& p2,
    const math::TVec2<scalar_t>& c) {
  return (
      (((p2.x >= c.x) && (c.x >= p1.x)) || ((p2.x <= c.x) && (c.x <= p1.x))) &&
      (((p2.y >= c.y) && (c.y >= p1.y)) || ((p2.y <= c.y) && (c.y <= p1.y))));
}

template <typename scalar_t>
__device__ inline math::TVec2<scalar_t>
get_cross_point(scalar_t a1, scalar_t b1, scalar_t c1, scalar_t a2, scalar_t b2, scalar_t c2) {
  scalar_t d = a1 * b2 - a2 * b1;
  if (d == scalar_t(0)) {
    return math::TVec2<scalar_t>{std::numeric_limits<scalar_t>().max()};
  }
  return math::TVec2<scalar_t>{(b1 * c2 - b2 * c1) / d, (a2 * c1 - a1 * c2) / d};
}

template <typename scalar_t>
__device__ inline math::TVec2<scalar_t> get_cross_point(
    scalar_t a1,
    scalar_t b1,
    scalar_t c1,
    const math::TVec2<scalar_t>& p1,
    const math::TVec2<scalar_t>& p2) {
  scalar_t a2 = 1e16;
  scalar_t b2 = 1e16;
  scalar_t c2 = 1e16;
  get_line(p1, p2, a2, b2, c2);
  scalar_t d = a1 * b2 - a2 * b1;
  if (d == scalar_t(0)) {
    return math::TVec2<scalar_t>{std::numeric_limits<scalar_t>().max()};
  }
  return math::TVec2<scalar_t>{(b1 * c2 - b2 * c1) / d, (a2 * c1 - a1 * c2) / d};
}

template <typename scalar_t>
__device__ inline bool is_crossing_dimond(
    const math::TVec2<scalar_t>& p1,
    const math::TVec2<scalar_t>& p2,
    const math::TVec2<scalar_t>& p) {
  scalar_t a0 = 1e16;
  scalar_t b0 = 1e16;
  scalar_t c0 = 1e16;
  get_line(p1, p2, a0, b0, c0);
  bool intersecting = false;
  {
    math::TVec2<scalar_t> s0 = {p.x, p.y - scalar_t(0.5)};
    math::TVec2<scalar_t> s1 = {p.x + scalar_t(0.5), p.y};
    auto c = get_cross_point(a0, b0, c0, s0, s1);
    intersecting |=
        is_point_in_segment<scalar_t>(s0, s1, c) && is_point_in_segment<scalar_t>(p1, p2, c);
  }
  {
    math::TVec2<scalar_t> s0 = {p.x + scalar_t(0.5), p.y};
    math::TVec2<scalar_t> s1 = {p.x, p.y + scalar_t(0.5)};
    auto c = get_cross_point(a0, b0, c0, s0, s1);
    intersecting |=
        is_point_in_segment<scalar_t>(s0, s1, c) && is_point_in_segment<scalar_t>(p1, p2, c);
  }
  {
    math::TVec2<scalar_t> s0 = {p.x, p.y + scalar_t(0.5)};
    math::TVec2<scalar_t> s1 = {p.x - scalar_t(0.5), p.y};
    auto c = get_cross_point(a0, b0, c0, s0, s1);
    intersecting |=
        is_point_in_segment<scalar_t>(s0, s1, c) && is_point_in_segment<scalar_t>(p1, p2, c);
  }
  {
    math::TVec2<scalar_t> s0 = {p.x - scalar_t(0.5), p.y};
    math::TVec2<scalar_t> s1 = {p.x, p.y - scalar_t(0.5)};
    auto c = get_cross_point(a0, b0, c0, s0, s1);
    intersecting |=
        is_point_in_segment<scalar_t>(s0, s1, c) && is_point_in_segment<scalar_t>(p1, p2, c);
  }
  return intersecting;
}

template <typename scalar_t, typename index_t>
__global__ void rasterize_lines_kernel(
    const index_t nthreads,
    TensorInfo<scalar_t, index_t> v,
    TensorInfo<int32_t, index_t> vi,
    TensorInfo<int64_t, index_t> packed_index_depth_img) {
  typedef typename math::TVec2<scalar_t> scalar2_t;
  typedef typename math::TVec3<scalar_t> scalar3_t;
  typedef typename math::TVec4<scalar_t> scalar4_t;

  const index_t H = packed_index_depth_img.sizes[1];
  const index_t W = packed_index_depth_img.sizes[2];
  const index_t V = v.sizes[1];
  const index_t n_prim = vi.sizes[1];

  const index_t index_sN = packed_index_depth_img.strides[0];
  const index_t index_sH = packed_index_depth_img.strides[1];
  const index_t index_sW = packed_index_depth_img.strides[2];

  const index_t v_sN = v.strides[0];
  const index_t v_sV = v.strides[1];
  const index_t v_sC = v.strides[2];

  const index_t vi_sN = vi.strides[0];
  const index_t vi_sF = vi.strides[1];
  const index_t vi_sI = vi.strides[2];

  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const index_t n = index / n_prim;
    const index_t id = index % n_prim;

    const int32_t* __restrict vi_ptr = vi.data + vi_sN * n + vi_sF * id;
    const int32_t flag = (int32_t)((((uint32_t)vi_ptr[vi_sI * 0] & 0xF0000000U)) >> 28U);
    const int32_t vi_0 = (int32_t)(((uint32_t)vi_ptr[vi_sI * 0]) & 0x0FFFFFFFU);
    const int32_t vi_1 = vi_ptr[vi_sI * 1];
    const int32_t vi_2 = vi_ptr[vi_sI * 2];

    // Skip degenerate triangles. Useful for padding of vi buffer for batched rasterization
    bool triangle_is_degenerate = (vi_0 == vi_1) && (vi_1 == vi_2);

    const bool edge_0_visible = (flag & 0b00000001) != 0;
    const bool edge_1_visible = (flag & 0b00000010) != 0;
    const bool edge_2_visible = (flag & 0b00000100) != 0;

    assert(vi_0 < V && vi_1 < V && vi_2 < V);

    const scalar_t* __restrict v_ptr = v.data + n * v_sN;
    const scalar2_t p_0 = {v_ptr[v_sV * vi_0 + v_sC * 0], v_ptr[v_sV * vi_0 + v_sC * 1]};
    const scalar2_t p_1 = {v_ptr[v_sV * vi_1 + v_sC * 0], v_ptr[v_sV * vi_1 + v_sC * 1]};
    const scalar2_t p_2 = {v_ptr[v_sV * vi_2 + v_sC * 0], v_ptr[v_sV * vi_2 + v_sC * 1]};

    const scalar3_t p_012_z = {
        v_ptr[v_sV * vi_0 + v_sC * 2],
        v_ptr[v_sV * vi_1 + v_sC * 2],
        v_ptr[v_sV * vi_2 + v_sC * 2]};

    const scalar2_t min_p = math::min(math::min(p_0, p_1), p_2);
    const scalar2_t max_p = math::max(math::max(p_0, p_1), p_2);

    const bool all_z_greater_0 = math::all_greater(p_012_z, {1e-8f, 1e-8f, 1e-8f});
    const bool in_canvas = math::all_less_or_eq(min_p, {(scalar_t)(W - 1), (scalar_t)(H - 1)}) &&
        math::all_greater(max_p, {0.f, 0.f});

    if (all_z_greater_0 && in_canvas && ~triangle_is_degenerate) {
      const scalar2_t v_01 = p_1 - p_0;
      const scalar2_t v_02 = p_2 - p_0;
      const scalar2_t v_12 = p_2 - p_1;

      const scalar_t denominator = v_01.x * v_02.y - v_01.y * v_02.x;

      if (denominator != 0.f) {
        // Compute triangle bounds with extra border.
        int min_x = max(1, int(min_p.x) - 2);
        int min_y = max(1, int(min_p.y) - 2);

        int max_x = min((int)W - 2, int(max_p.x) + 2);
        int max_y = min((int)H - 2, int(max_p.y) + 2);

        // Loop over pixels inside triangle bbox.
        for (int y = min_y; y <= max_y; ++y) {
          for (int x = min_x; x <= max_x; ++x) {
            const scalar2_t p = {(scalar_t)x, (scalar_t)y};

            const scalar2_t vp0p = p - p_0;
            const scalar2_t vp1p = p - p_1;

            bool intersecting = false;
            intersecting |= is_crossing_dimond<scalar_t>(p_0, p_1, p) && edge_0_visible;
            intersecting |= is_crossing_dimond<scalar_t>(p_1, p_2, p) && edge_1_visible;
            intersecting |= is_crossing_dimond<scalar_t>(p_0, p_2, p) && edge_2_visible;

            scalar3_t bary = scalar3_t({
                vp1p.y * v_12.x - vp1p.x * v_12.y,
                vp0p.x * v_02.y - vp0p.y * v_02.x,
                vp0p.y * v_01.x - vp0p.x * v_01.y,
            });
            bary *= sign(denominator);

            const bool on_edge_or_inside = (bary.x >= 0.f) && (bary.y >= 0.f) && (bary.z >= 0.f);

            bool on_edge_0 = bary.x == 0.f;
            bool on_edge_1 = bary.y == 0.f;
            bool on_edge_2 = bary.z == 0.f;

            const bool is_top_left_0 = (denominator > 0)
                ? (v_12.y < 0.f || v_12.y == 0.0f && v_12.x > 0.f)
                : (v_12.y > 0.f || v_12.y == 0.0f && v_12.x < 0.f);
            const bool is_top_left_1 = (denominator > 0)
                ? (v_02.y > 0.f || v_02.y == 0.0f && v_02.x < 0.f)
                : (v_02.y < 0.f || v_02.y == 0.0f && v_02.x > 0.f);
            const bool is_top_left_2 = (denominator > 0)
                ? (v_01.y < 0.f || v_01.y == 0.0f && v_01.x > 0.f)
                : (v_01.y > 0.f || v_01.y == 0.0f && v_01.x < 0.f);

            const bool is_top_left_or_inside = on_edge_or_inside &&
                !(on_edge_0 && !is_top_left_0 || on_edge_1 && !is_top_left_1 ||
                  on_edge_2 && !is_top_left_2);

            if (is_top_left_or_inside || intersecting) {
              bary /= abs(denominator);
              bary = math::max(bary, scalar3_t{0, 0, 0});
              bary = math::min(bary, scalar3_t{1, 1, 1});
              bary = bary / math::sum(bary);

              // interpolate inverse depth linearly
              const scalar3_t d_inv = 1.0 / epsclamp(p_012_z);
              const scalar_t depth_inverse = dot(d_inv, bary);
              const scalar_t depth = 1.0f / epsclamp(depth_inverse);

              const unsigned long long packed_val =
                  (static_cast<unsigned long long>(__float_as_uint(depth)) << 32u) |
                  (intersecting ? static_cast<unsigned long long>(id) : 0xFFFFFFFFULL);
              atomicMin(
                  reinterpret_cast<unsigned long long*>(packed_index_depth_img.data) +
                      index_sN * n + index_sH * y + index_sW * x,
                  packed_val);
            }
          }
        }
      }
    }
  }
}

template <typename index_t>
__global__ void unpack_kernel(
    const index_t nthreads,
    TensorInfo<int64_t, index_t> packed_index_depth_img,
    TensorInfo<float, index_t> depth_img,
    TensorInfo<int32_t, index_t> index_img) {
  CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
    const unsigned long long int pv =
        reinterpret_cast<unsigned long long int*>(packed_index_depth_img.data)[index];
    const auto depth_uint = static_cast<uint32_t>(pv >> 32);
    depth_img.data[index] = depth_uint == 0xFFFFFFFF ? 0.0f : __uint_as_float(depth_uint);
    reinterpret_cast<uint32_t*>(index_img.data)[index] = static_cast<uint32_t>(pv & 0xFFFFFFFF);
  }
}

std::vector<torch::Tensor> rasterize_cuda(
    const torch::Tensor& v,
    const torch::Tensor& vi,
    int64_t height,
    int64_t width,
    bool wireframe) {
  TORCH_CHECK(v.defined() && vi.defined(), "rasterize(): expected all inputs to be defined");
  auto v_opt = v.options();
  auto vi_opt = vi.options();
  TORCH_CHECK(
      (v.device() == vi.device()) && (v.is_cuda()),
      "rasterize(): expected all inputs to be on same cuda device");
  TORCH_CHECK(
      v.is_floating_point(),
      "rasterize(): expected v to have floating point type, but v has ",
      v.dtype());
  TORCH_CHECK(
      vi.dtype() == torch::kInt32,
      "rasterize(): expected vi to have int32 type, but vi has ",
      vi.dtype());
  TORCH_CHECK(
      v.layout() == torch::kStrided && vi.layout() == torch::kStrided,
      "rasterize(): expected all inputs to have torch.strided layout");
  TORCH_CHECK(
      (v.dim() == 3) && (vi.dim() == 3),
      "rasterize(): expected v.ndim == 3, vi.ndim == 3, "
      "but got v with sizes ",
      v.sizes(),
      " and vi with sizes ",
      vi.sizes());
  TORCH_CHECK(
      v.size(2) == 3 && vi.size(2) == 3,
      "rasterize(): expected third dim of v to be of size 3, and last dim of vi to be of size 3, but got ",
      v.size(2),
      " in the third dim of v, and ",
      vi.size(2),
      " in the last dim of vi");
  TORCH_CHECK(
      (vi.size(0) == v.size(0)),
      "rasterize(): expected first dim of vi to match first dim of v, but got ",
      v.size(0),
      " in first dim of v, and ",
      vi.size(0),
      " in the first dim of vi");
  TORCH_CHECK(
      v.size(1) < 0x10000000U,
      "rasterize(): expected second dim of v to be less or eual to 268435456, but got ",
      v.size(1));
  TORCH_CHECK(
      height > 0 && width > 0,
      "rasterize(): both height and width have to be greater than zero, but got height: ",
      height,
      ", and width: ",
      width);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(v));
  auto stream = at::cuda::getCurrentCUDAStream();

  auto N = v.size(0);
  auto T = vi.size(1);
  auto H = height;
  auto W = width;
  const auto count_rasterize = N * T;
  const auto count_unpack = N * H * W;

  auto packed_index_depth_img = at::empty({N, H, W}, v.options().dtype(torch::kInt64));
  auto depth_img = at::empty({N, H, W}, v.options().dtype(torch::kFloat32));
  auto index_img = at::empty({N, H, W}, v.options().dtype(torch::kInt32));

  hipMemsetAsync(
      packed_index_depth_img.data_ptr(),
      0xFF,
      N * H * W * torch::elementSize(torch::kInt64),
      stream);

  // rasterize
  if (count_rasterize > 0) {
    AT_DISPATCH_FLOATING_TYPES(v.scalar_type(), "rasterize_kernel", [&] {
      if (at::native::canUse32BitIndexMath(v) && at::native::canUse32BitIndexMath(vi) &&
          at::native::canUse32BitIndexMath(packed_index_depth_img)) {
        typedef int index_type;

        if (wireframe) {
          rasterize_lines_kernel<scalar_t, index_type>
              <<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
                  static_cast<index_type>(count_rasterize),
                  getTensorInfo<scalar_t, index_type>(v),
                  getTensorInfo<int32_t, index_type>(vi),
                  getTensorInfo<int64_t, index_type>(packed_index_depth_img));
        } else {
          rasterize_kernel<scalar_t, index_type>
              <<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
                  static_cast<index_type>(count_rasterize),
                  getTensorInfo<scalar_t, index_type>(v),
                  getTensorInfo<int32_t, index_type>(vi),
                  getTensorInfo<int64_t, index_type>(packed_index_depth_img));
        }
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        typedef int64_t index_type;

        if (wireframe) {
          rasterize_lines_kernel<scalar_t, index_type>
              <<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
                  static_cast<index_type>(count_rasterize),
                  getTensorInfo<scalar_t, index_type>(v),
                  getTensorInfo<int32_t, index_type>(vi),
                  getTensorInfo<int64_t, index_type>(packed_index_depth_img));
        } else {
          rasterize_kernel<scalar_t, index_type>
              <<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
                  static_cast<index_type>(count_rasterize),
                  getTensorInfo<scalar_t, index_type>(v),
                  getTensorInfo<int32_t, index_type>(vi),
                  getTensorInfo<int64_t, index_type>(packed_index_depth_img));
        }

        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }

  // unpack
  if (count_unpack > 0) {
    if (at::native::canUse32BitIndexMath(packed_index_depth_img) &&
        at::native::canUse32BitIndexMath(depth_img) &&
        at::native::canUse32BitIndexMath(index_img)) {
      typedef int index_type;

      unpack_kernel<index_type><<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
          static_cast<index_type>(count_unpack),
          getTensorInfo<int64_t, index_type>(packed_index_depth_img),
          getTensorInfo<float, index_type>(depth_img),
          getTensorInfo<int32_t, index_type>(index_img));
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      typedef int64_t index_type;

      unpack_kernel<index_type><<<GET_BLOCKS(count_rasterize, 256), 256, 0, stream>>>(
          static_cast<index_type>(count_unpack),
          getTensorInfo<int64_t, index_type>(packed_index_depth_img),
          getTensorInfo<float, index_type>(depth_img),
          getTensorInfo<int32_t, index_type>(index_img));
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }

  return {depth_img, index_img};
}
